#include "hip/hip_runtime.h"
#include "../functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void divergent_func_opt(int *ret) {
  int tx = threadIdx.x;
  int m = 128 * tx;
  int o = 128 * (tx + 2);

  bool p1 = tx < 16;

  // Predicate the arguments
  int j = o;
  if (p1)
    j = m;

  // Single function call, allowing the call to be done in lock-step
  FUNC(j);
  ret[tx] = m + o;
  return;
}

int main() {
  printf("Opt\n");

  int *out, *d_out;
  // Allocate host memory
  out = (int *)malloc(sizeof(int) * NUM_THREADS_PER_WARP);
  // Allocate device memory
  hipMalloc((void **)&d_out, sizeof(int) * NUM_THREADS_PER_WARP);
  // Executing kernel
  divergent_func_opt<<<NUM_WARPS, NUM_THREADS_PER_WARP>>>(d_out);
  hipMemcpy(out, d_out, sizeof(int) * NUM_THREADS_PER_WARP,
             hipMemcpyDeviceToHost);
  for (int i = 0; i < NUM_THREADS_PER_WARP; i++)
    printf("flat: out[%d] = %d\n", i, out[i]);
  printf("\n");

  printf("\n Error msg: %s\n", hipGetErrorString(hipGetLastError()));

  // Deallocate device memory
  hipFree(d_out);
  // Deallocate hostmemory
  free(out);
}
