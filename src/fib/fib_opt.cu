#include "hip/hip_runtime.h"
#include "../functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

__global__ void divergent_func_opt(int *ret) {
  int tx = threadIdx.x;
  int r = 3;
  int z = 4;
  int m = 127 * tx;
  int o = 127 * (tx + 2);

  // Max 27, else errors out
  m %= MOD;
  o %= MOD;

  bool p1 = tx < 16;

  int j = o;
  if (p1)
    j = m;

  int *t = &z;
  if (p1)
    t = &r;

  recursive_fibonacci(j, t);
  ret[tx] = r + z;
  return;
}

int main() {
  printf("Opt\n");
  int *out, *d_out;
  // Allocate host memory
  out = (int *)malloc(sizeof(int) * NUM_THREADS_PER_WARP);
  // Allocate device memory
  hipMalloc((void **)&d_out, sizeof(int) * NUM_THREADS_PER_WARP);
  // Executing kernel
  divergent_func_opt<<<NUM_WARPS, NUM_THREADS_PER_WARP>>>(d_out);
  hipMemcpy(out, d_out, sizeof(int) * NUM_THREADS_PER_WARP,
             hipMemcpyDeviceToHost);
  for (int i = 0; i < NUM_THREADS_PER_WARP; i++)
    printf("flat: out[%d] = %d\n", i, out[i]);
  printf("\n");

  printf("\n Error msg: %s\n", hipGetErrorString(hipGetLastError()));

  // Deallocate device memory
  hipFree(d_out);
  // Deallocate hostmemory
  free(out);
}
