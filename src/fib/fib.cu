#include "hip/hip_runtime.h"
#include "../functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

__global__ void divergent_func(int *ret) {
  int tx = threadIdx.x;
  int r = 3;
  int z = 4;
  int m = 127 * tx;
  int o = 127 * (tx + 2);

  // Max 27, else errors out
  m %= MOD;
  o %= MOD;
  bool p1 = tx < 16;
  if (p1) {
    recursive_fibonacci(m, &r);
  } else {
    recursive_fibonacci(o, &z);
  }
  // so DCE doesn't eliminate stuff
  ret[tx] = r + z;
  return;
}

int main() {
  printf("Base\n");
  int *out, *d_out;
  // Allocate host memory
  out = (int *)malloc(sizeof(int) * NUM_THREADS_PER_WARP);
  // Allocate device memory
  hipMalloc((void **)&d_out, sizeof(int) * NUM_THREADS_PER_WARP);
  // Executing kernel
  divergent_func<<<NUM_WARPS, NUM_THREADS_PER_WARP>>>(d_out);
  hipMemcpy(out, d_out, sizeof(int) * NUM_THREADS_PER_WARP,
             hipMemcpyDeviceToHost);
  for (int i = 0; i < NUM_THREADS_PER_WARP; i++)
    printf("flat: out[%d] = %d\n", i, out[i]);
  printf("\n");

  printf("\n Error msg: %s\n", hipGetErrorString(hipGetLastError()));

  // Deallocate device memory
  hipFree(d_out);
  // Deallocate hostmemory
  free(out);
}
